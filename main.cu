#include "hip/hip_runtime.h"
#include <random>
#include <string>
#include <vector>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <omp.h>
#include "inc/main.hxx"

using namespace std;




#pragma region CONFIGURATION
#ifndef TYPE
/** Type of PageRank values. */
#define TYPE double
#endif
#ifndef MAX_THREADS
/** Maximum number of threads to use. */
#define MAX_THREADS 32
#endif
#ifndef REPEAT_BATCH
/** Number of times to repeat each batch. */
#define REPEAT_BATCH 5
#endif
#ifndef REPEAT_METHOD
/** Number of times to repeat each method. */
#define REPEAT_METHOD 1
#endif
#pragma endregion




#pragma region METHODS
#pragma region EXPERIMENTAL SETUP
/**
 * Run a function on each batch update, with a specified range of batch sizes.
 * @param x original graph
 * @param rnd random number generator
 * @param fn function to run on each batch update
 */
template <class G, class R, class F>
inline void runBatches(const G& x, R& rnd, F fn) {
  auto  fl = [](auto u) { return true; };
  double d = BATCH_DELETIONS_BEGIN;
  double i = BATCH_INSERTIONS_BEGIN;
  while (true) {
    for (int r=0; r<REPEAT_BATCH; ++r) {
      auto y  = duplicate(x);
      auto deletions  = removeRandomEdges(y, rnd, size_t(d * x.size() + 0.5), 1, x.span()-1);
      auto insertions = addRandomEdges   (y, rnd, size_t(i * x.size() + 0.5), 1, x.span()-1, None());
      addSelfLoopsOmpU(y, None(), fl);
      auto yt = transposeWithDegreeOmp(y);
      fn(y, yt, d, deletions, i, insertions);
    }
    if (d>=BATCH_DELETIONS_END && i>=BATCH_INSERTIONS_END) break;
    d BATCH_DELETIONS_STEP;
    i BATCH_INSERTIONS_STEP;
    d = min(d, double(BATCH_DELETIONS_END));
    i = min(i, double(BATCH_INSERTIONS_END));
  }
}


/**
 * Run a function on each number of threads, with a specified range of thread counts.
 * @param fn function to run on each number of threads
 */
template <class F>
inline void runThreads(F fn) {
  for (int t=NUM_THREADS_BEGIN; t<=NUM_THREADS_END; t NUM_THREADS_STEP) {
    omp_set_num_threads(t);
    fn(t);
    omp_set_num_threads(MAX_THREADS);
  }
}
#pragma endregion




#pragma region PERFORM EXPERIMENT
/**
 * Perform the experiment.
 * @param x original graph
 * @param xt transposed graph with degree
 */
template <class G, class H>
inline void runExperiment(const G& x, const H& xt) {
  using  K = typename G::key_type;
  using  V = TYPE;
  vector<V> *init = nullptr;
  random_device dev;
  default_random_engine rnd(dev());
  int repeat = REPEAT_METHOD;
  // Follow a specific result logging format, which can be easily parsed later.
  auto glog  = [&](const auto& ans, const auto& ref, const char *technique, auto deletionsf, auto insertionsf, int numThreads) {
    auto err = liNormDeltaOmp(ans.ranks, ref.ranks);
    printf(
      "{-%.3e/+%.3e batchf, %03d threads} -> {%09.1fms, %03d iter, %.2e err} %s\n",
      deletionsf, insertionsf, numThreads, ans.time, ans.iterations, err, technique
    );
  };
  // Get ranks of vertices on original graph (static).
  auto r0   = pagerankStaticOmp(xt, init, {1, 1e-100});
  // Get ranks of vertices on updated graph (dynamic).
  runBatches(x, rnd, [&](const auto& y, const auto& yt, double deletionsf, const auto& deletions, double insertionsf, const auto& insertions) {
    runThreads([&](int numThreads) {
      auto flog = [&](const auto& ans, const auto& ref, const char *technique) {
        glog(ans, ref, technique, deletionsf, insertionsf, numThreads);
      };
      auto s0 = pagerankStaticOmp(yt, init, {1, 1e-100});
      // Find multi-threaded OpenMP-based Static PageRank (synchronous, no dead ends).
      auto a0 = pagerankStaticOmp(yt, init, {repeat});
      flog(a0, s0, "pagerankStaticOmp");
      auto b0 = pagerankStaticCuda(y, yt, init, {repeat});
      flog(b0, s0, "pagerankStaticCuda");
      // Find multi-threaded OpenMP-based Naive-dynamic PageRank (synchronous, no dead ends).
      auto a1 = pagerankStaticOmp(yt, &r0.ranks, {repeat});
      flog(a1, s0, "pagerankNaiveDynamicOmp");
      // auto b1 = pagerankStaticCuda(y, yt, &r0.ranks, {repeat});
      // flog(b1, s0, "pagerankNaiveDynamicCuda");
      // Find multi-threaded OpenMP-based Frontier-based Dynamic PageRank (synchronous, no dead ends).
      auto a2 = pagerankDynamicFrontierOmp(x, xt, y, yt, deletions, insertions, &r0.ranks, {repeat});
      flog(a2, s0, "pagerankDynamicFrontierOmp");
    });
  });
}


/**
 * Main function.
 * @param argc argument count
 * @param argv argument values
 * @returns zero on success, non-zero on failure
 */
int main(int argc, char **argv) {
  char *file = argv[1];
  omp_set_num_threads(MAX_THREADS);
  LOG("OMP_NUM_THREADS=%d\n", MAX_THREADS);
  LOG("Loading graph %s ...\n", file);
  DiGraph<uint32_t> x;
  readMtxOmpW(x, file); LOG(""); println(x);
  auto fl = [](auto u) { return true; };
  x = addSelfLoopsOmp(x, None(), fl);  LOG(""); print(x);  printf(" (selfLoopAllVertices)\n");
  auto xt = transposeWithDegreeOmp(x); LOG(""); print(xt); printf(" (transposeWithDegree)\n");
  runExperiment(x, xt);
  printf("\n");
  return 0;
}
#pragma endregion
#pragma endregion
