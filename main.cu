#include "hip/hip_runtime.h"
#include <cmath>
#include <string>
#include <sstream>
#include <cstdio>
#include <iostream>
#include <utility>
#include <algorithm>
#include "src/main.hxx"

using namespace std;




#define REPEAT 5

void runPagerankBatch(const string& data, bool show, int batch, int skip) {
  vector<float>  ranksOld, ranksAdj;
  vector<float> *initStatic  = nullptr;
  vector<float> *initDynamic = &ranksAdj;

  DiGraph<> x;
  stringstream s(data);
  auto ksOld = vertices(x);
  while(readSnapTemporal(x, s, batch)) {
    auto ks = vertices(x);
    auto xt = transposeWithDegree(x);
    ranksAdj.resize(x.span());

    // Find static pagerank of updated graph using nvGraph.
    auto a1 = pagerankNvgraph(xt, initStatic, {REPEAT});
    auto e1 = l1Norm(a1.ranks, a1.ranks);
    print(xt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankNvgraph [static]\n", a1.time, a1.iterations, e1);

    // Find static pagerank of updated graph using CUDA.
    auto a2 = pagerankCuda(xt, initStatic, {REPEAT});
    auto e2 = l1Norm(a2.ranks, a1.ranks);
    print(xt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankCuda [static]\n", a2.time, a2.iterations, e2);

    // Find dynamic pagerank of updated graph using nvGraph.
    adjustRanks(ranksAdj, ranksOld, ksOld, ks, 0.0f, float(ksOld.size())/ks.size(), 1.0f/ks.size());
    auto a3 = pagerankNvgraph(xt, initDynamic, {REPEAT});
    auto e3 = l1Norm(a3.ranks, a1.ranks);
    print(xt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankNvgraph [dynamic]\n", a3.time, a3.iterations, e3);

    // Find dynamic pagerank of updated graph using CUDA.
    auto a4 = pagerankCuda(xt, initDynamic, {REPEAT});
    auto e4 = l1Norm(a4.ranks, a1.ranks);
    print(xt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankCuda [dynamic]\n", a4.time, a4.iterations, e4);

    // Skip some edges (to speed up execution)
    if (skip) {
      if (!readSnapTemporal(x, s, skip)) break;
      ks = vertices(x);
      xt = transposeWithDegree(x);
      a1 = pagerankCuda(xt, initStatic);
    }

    ksOld = move(ks);
    ranksOld = move(a1.ranks);
  }
}


void runPagerank(const string& data, bool show) {
  int M = countLines(data), steps = 100;
  printf("Temporal edges: %d\n", M);
  for (int batch=1, i=0; batch<M; batch*=i&1? 2:5, i++) {
    int skip = max(M/steps - batch, 0);
    printf("\n# Batch size %.0e\n", (double) batch);
    runPagerankBatch(data, show, batch, skip);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  bool  show = argc > 2;
  printf("Using graph %s ...\n", file);
  string d = readFile(file);
  runPagerank(d, show);
  printf("\n");
  return 0;
}
