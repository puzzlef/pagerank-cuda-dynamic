#include "hip/hip_runtime.h"
#include <cmath>
#include <string>
#include <sstream>
#include <cstdio>
#include <iostream>
#include <utility>
#include <algorithm>
#include "src/main.hxx"

using namespace std;




#define REPEAT 5

void runPagerankBatch(const string& data, bool show, int batch, int skip) {
  vector<float>  ranksOld, ranksAdj;
  vector<float> *initStatic  = nullptr;
  vector<float> *initDynamic = &ranksAdj;

  DiGraph<> x;
  stringstream s(data);
  auto ksOld = vertices(x);
  while(readSnapTemporal(x, s, batch)) {
    auto ks = vertices(x);
    auto xt = transposeWithDegree(x);
    ranksAdj.resize(x.span());

    // Find static pagerank of updated graph.
    auto a1 = pagerankCuda(xt, initStatic, {REPEAT});
    auto e1 = l1Norm(a1.ranks, a1.ranks);
    print(xt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankStatic\n", a1.time, a1.iterations, e1);

    // Find dynamic pagerank, scaling old vertices, and using 1/N for new vertices.
    adjustRanks(ranksAdj, ranksOld, ksOld, ks, 0.0f, float(ksOld.size())/ks.size(), 1.0f/ks.size());
    auto a2 = pagerankCuda(xt, initDynamic, {REPEAT});
    auto e2 = l1Norm(a2.ranks, a1.ranks);
    print(xt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankDynamic\n", a2.time, a2.iterations, e2);

    // Skip some edges (to speed up execution)
    if (skip) {
      if (!readSnapTemporal(x, s, skip)) break;
      ks = vertices(x);
      xt = transposeWithDegree(x);
      a1 = pagerankCuda(xt, initStatic);
    }

    ksOld = move(ks);
    ranksOld = move(a1.ranks);
  }
}


void runPagerank(const string& data, bool show) {
  int M = countLines(data), steps = 100;
  printf("Temporal edges: %d\n", M);
  for (int batch=1, i=0; batch<M; batch*=i&1? 2:5, i++) {
    int skip = max(M/steps - batch, 0);
    printf("\n# Batch size %.0e\n", (double) batch);
    runPagerankBatch(data, show, batch, skip);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  bool  show = argc > 2;
  printf("Using graph %s ...\n", file);
  string d = readFile(file);
  runPagerank(d, show);
  printf("\n");
  return 0;
}
